#include "hip/hip_runtime.h"
/***************************************************************************
 *   Copyright (C) 2004 by Mario Juric                                     *
 *   mjuric@astro.Princeton.EDU                                            *
 *                                                                         *
 *   This program is free software; you can redistribute it and/or modify  *
 *   it under the terms of the GNU General Public License as published by  *
 *   the Free Software Foundation; either version 2 of the License, or     *
 *   (at your option) any later version.                                   *
 *                                                                         *
 *   This program is distributed in the hope that it will be useful,       *
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of        *
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the         *
 *   GNU General Public License for more details.                          *
 *                                                                         *
 *   You should have received a copy of the GNU General Public License     *
 *   along with this program; if not, write to the                         *
 *   Free Software Foundation, Inc.,                                       *
 *   59 Temple Place - Suite 330, Boston, MA  02111-1307, USA.             *
 ***************************************************************************/

#ifndef __CONFIG_H
#define __CONFIG_H
#include "config.h"
#endif

#include <stdint.h>
#include <math.h>

#include <astro/constants.h>

#include "simulate_base.h"
#include "column.h"
#include "gpu.h"

namespace ct = column_types;
KERNEL(
	ks,
	os_FeH_kernel(otable_ks ks, os_FeH_data par, gpu_rng_t rng, ct::cint::gpu_t comp, ct::cfloat::gpu_t XYZ, ct::cfloat::gpu_t FeH),
	os_FeH_kernel,
	(ks, par, rng, comp, XYZ, FeH)
)
{
	uint32_t row = ks.row();
	if(row == (uint32_t)(-1)) { return; }
	rng.load(ks);

	float feh;
	switch(comp[row])
	{
		case 0: // BahcallSoneira_model::THIN:
		case 1: // BahcallSoneira_model::THICK:
		{
			// choose the gaussian to draw from
			float p = rng.uniform()*(par.A[0]+par.A[1]);
			int i = p < par.A[0] ? 0 : 1;

			// calculate mean
			float muD = par.muInf + par.DeltaMu*exp(-fabs(XYZ(row, 2))/par.Hmu);		// Bond et al. A2
			float aZ = muD - 0.067f;

			// draw
			feh = rng.gaussian(par.sigma[i]) + aZ + par.offs[i];
		} break;
		case 2: //BahcallSoneira_model::HALO:
			feh = par.offs[2] + rng.gaussian(par.sigma[2]);
			break;
		default:
			//THROW(ENotImplemented, "We should have never gotten here");
			feh = -9999.f;
			break;
	}
	FeH[row] = feh;
}


// equgal - Equatorial to Galactic coordinates
using namespace peyton;
typedef double Radians;
static const double angp = ctn::d2r * 192.859508333; //  12h 51m 26.282s (J2000)
static const double dngp = ctn::d2r * 27.128336111;  // +27d 07' 42.01" (J2000)
static const double l0 = ctn::d2r * 32.932;
static const double ce = 0.88998740217659689; // cos(dngp)
static const double se = 0.45598511375586859; // sin(dngp)

inline __device__ double2 galequ(const double2 lb)
{
	const double cb = cos(lb.y);
	const double sb = sin(lb.y);
	const double cl = cos(lb.x-l0);
	const double sl = sin(lb.x-l0);

//	// TODO: These should be precomputed constants
//	const double ce = cos(dngp);
//	const double se = sin(dngp);

	double2 r;
	r.x = atan2(
			cb*cl,
			sb*ce-cb*se*sl
		) + angp;
	r.y = asin(cb*ce*sl + sb*se);

	while(r.x < 0.) { r.x += ctn::pi2; }
	return r;
}

KERNEL(
	ks,
	os_gal2other_kernel(otable_ks ks, int coordsys, ct::cdouble::gpu_t lb0, ct::cdouble::gpu_t out),
	os_gal2other_kernel,
	(ks, coordsys, lb0, out)
)
{
	uint32_t row = ks.row();
	if(row == (uint32_t)(-1)) { return; }

	double2 lb, ret;

	// convert to radians
	lb.x = lb0(row, 0) * ctn::d2r;
	lb.y = lb0(row, 1) * ctn::d2r;

	// rotate to output coordinate system
	switch(coordsys)
	{
	case EQU:
		ret = galequ(lb);
		break;
	default:
		ret.x = ret.y = -9999.;
		break;
	}

	// convert to degrees
	out(row, 0) = ret.x / ctn::d2r;
	out(row, 1) = ret.y / ctn::d2r;
}
